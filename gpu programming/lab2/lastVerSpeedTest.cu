#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <malloc.h>
#include <math.h>
#include "hip/hip_runtime.h"

__global__ void Sample(int *masA, int *masB){
	masA[threadIdx.x+blockDim.x*blockIdx.x]+=masB[threadIdx.x+blockDim.x*blockIdx.x];
}

__global__ void CreateMas(int *masA, int *masB){
	masA[threadIdx.x+blockDim.x*blockIdx.x]=(int)(2*(threadIdx.x+blockDim.x*blockIdx.x));
	masB[threadIdx.x+blockDim.x*blockIdx.x]=(int)(2*(threadIdx.x+blockDim.x*blockIdx.x)+1);
}

int main(int argc, char **argv){
	if(argc<2){
		fprintf(stdout," Need 4 argum - prog nCoreInBlock.\n");
		return 4;
	}
	int nn=300000000, nCoreInBlock=atoi(argv[1]);
	int nBlock=ceil(nn/(float)nCoreInBlock), i=0;
	int *mas, *masA, *masB;
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	mas=(int*)calloc(nn,sizeof(int));
	hipMalloc((void**)&masA,nn*sizeof(int));
	hipMalloc((void**)&masB,nn*sizeof(int));
	
	//CreateMas<<<dim3(nBlock),dim3(nCoreInBlock)>>>(masA,masB);
	//cudaDeviceSynchronize();
	hipEventRecord(start,0);
	CreateMas<<<dim3(nBlock),dim3(nCoreInBlock)>>>(masA,masB);
	hipDeviceSynchronize();
	Sample<<<dim3(nBlock),dim3(nCoreInBlock)>>>(masA,masB);
	hipDeviceSynchronize();
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipMemcpy(mas,masA,nn*sizeof(float),hipMemcpyDeviceToHost);
	hipEventElapsedTime(&elapsedTime,start,stop);
	
	fprintf(stdout," Time - %g\n",elapsedTime);
	int shag=nn/8;
	if(shag==0) shag=2;
	for(i=0;i<nn;i+=shag) fprintf(stdout,"%d\t%d\n",i,mas[i]);
	
	free(mas);
	hipFree(masA);
	hipFree(masB);
	return 0;
}
