#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N (1024*1024)
#define FULL_DATA (N*20)

__global__ void kernel(int *a, int *b, int *c){
		int temp=threadIdx.x+blockIdx.x*blockDim.x;
		if(temp<N){
			int idx1=(temp+1)%256;
			int idx2=(temp+2)%256;
			float as=(a[temp]+a[idx1]+a[idx2])/3.0f;
			float bs=(b[temp]+b[idx1]+b[idx2])/3.0f;
			c[temp]=(as+bs)/2;
		}
}

int main(){
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop,whichDevice);
	if(!prop.deviceOverlap){
		printf("Device does not support overlapping\n");
		return -1;
	}
	int *hostA, *hostB, *hostC;
	int *devA0, *devA1, *devB0, *devB1, *devC0, *devC1;
	hipMalloc((void**)&devA0,N*sizeof(int));
	hipMalloc((void**)&devA1,N*sizeof(int));
	hipMalloc((void**)&devB0,N*sizeof(int));
	hipMalloc((void**)&devB1,N*sizeof(int));
	hipMalloc((void**)&devC0,N*sizeof(int));
	hipMalloc((void**)&devC1,N*sizeof(int));
	hipHostMalloc((void**)&hostA,FULL_DATA*sizeof(int));
	hipHostMalloc((void**)&hostB,FULL_DATA*sizeof(int));
	hipHostMalloc((void**)&hostC,FULL_DATA*sizeof(int));
	for(int i=0;i<FULL_DATA;i+=1){
		hostA[i]=rand();
		hostB[i]=rand();
	}
	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	for(int i=0;i<FULL_DATA;i+=N*2){
		hipMemcpyAsync(devA0,hostA+i,N*sizeof(int),hipMemcpyHostToDevice,stream0);
		hipMemcpyAsync(devA1,hostA+i+N,N*sizeof(int),hipMemcpyHostToDevice,stream1);
		hipMemcpyAsync(devB0,hostB+i,N*sizeof(int),hipMemcpyHostToDevice,stream0);
		hipMemcpyAsync(devB1,hostB+i+N,N*sizeof(int),hipMemcpyHostToDevice,stream1);
		kernel<<<dim3(N/256),dim3(256),0,stream0>>>(devA0,devB0,devC0);
		kernel<<<dim3(N/256),dim3(256),0,stream1>>>(devA1,devB1,devC1);
		hipMemcpyAsync(hostC+i,devC0,N*sizeof(int),hipMemcpyDeviceToHost,stream0);
		hipMemcpyAsync(hostC+i+N,devC1,N*sizeof(int),hipMemcpyDeviceToHost,stream1);
	}
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);
	hipFree(devA0);
	hipFree(devA1);
	hipFree(devB0);
	hipFree(devB1);
	hipFree(devC0);
	hipFree(devC1);
	hipHostFree(hostA);
	hipHostFree(hostB);
	hipHostFree(hostC);
	return 0;
}
