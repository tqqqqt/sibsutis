#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <math.h>


#define COEF 48
#define VERTCOUNT COEF*COEF*2
#define M_PI 3.141592
#define RADIUS 10.0f
#define FGSIZE 20
#define MFGSIZE FGSIZE/2
#define FGSHIFT FGSIZE/2
#define THREADSPERBLOCK 256
#define IMIN(A,B) (A<B?A:B)
#define BLOCKGRID IMIN(32,(VERTCOUNT+THREADSPERBLOCK-1)/THREADSPERBLOCK)

#define CUDA_CHECK_RETURN(value) {\
	hipError_t _m_cudaStat=value;\
	if(_m_cudaStat!=hipSuccess){\
		fprintf(stderr,"Error %s at line %d in file %s\n",hipGetErrorString(_m_cudaStat),__LINE__,__FILE__);\
		exit(1);\
	} }

struct Vertex {
	float x, y, z;
};

struct FuncBase {
	float x, y, z;
	float res;
};


float Function(float vx, float vy, float vz) { 
	return (0.5 * sqrtf(15.0 / M_PI)) * (0.5 * sqrtf(15.0 / M_PI)) * vz * vz * vy * vy * sqrtf(1.0 - vz * vz / RADIUS / RADIUS) / RADIUS / RADIUS / RADIUS / RADIUS;
}

float check(Vertex* v) {
	float sum = 0;
	int i = 0;
	for (i = 0;i < VERTCOUNT;i++) {
		sum += Function(v[i].x,v[i].y,v[i].z);
	}
	return sum;
}

void InitVertices(struct Vertex *vertex) {
	struct Vertex *tempVert = (struct Vertex*)malloc(VERTCOUNT * sizeof(struct Vertex));
	int i = 0, iphi = 0, ipsi = 0;
	for (iphi = 0;iphi < 2 * COEF;++iphi) {
		for (ipsi = 0;ipsi < COEF;++ipsi, ++i) {
			float phi = iphi * M_PI / COEF;
			float psi = ipsi * M_PI / COEF;
			tempVert[i].x = RADIUS * sinf(psi) * cosf(phi);
			tempVert[i].y = RADIUS * sinf(psi) * sinf(phi);
			tempVert[i].z = RADIUS * cosf(psi);
		}
	}
	printf("\n Sumcheck = %f\n", check(tempVert) * M_PI * M_PI / COEF / COEF);
	hipMemcpy(vertex, tempVert, VERTCOUNT * sizeof(Vertex), hipMemcpyHostToDevice);
	free(tempVert);
}

__device__ void StepFixBound(float* cord) {
	if (*cord < 0-FGSHIFT) {
		*cord = 0-FGSHIFT;
		return;
	}
	if (*cord > (FGSIZE-1)-FGSHIFT) {
		*cord = (FGSIZE - 1) - FGSHIFT;
		return;
	}
}

__device__ float StepInter(struct FuncBase *mas, float x, float y, float z) {
	int i = 0, tempNum=-1;
	float tempX = floor(x), tempY = floor(y), tempZ = floor(z);
	StepFixBound(&tempX);
	StepFixBound(&tempY);
	StepFixBound(&tempZ);
	for (i = 0;i < FGSIZE * FGSIZE * FGSIZE;i++) {
		if (mas[i].x > tempX || mas[i].y > tempY || mas[i].z > tempZ) continue;
		if (tempNum == -1) tempNum = i;		
		if (mas[i].x >= mas[tempNum].x && mas[i].y >= mas[tempNum].y && mas[i].z >= mas[tempNum].z) tempNum = i;
	}
	if (tempNum == -1) return 0.0;
	return mas[tempNum].res;
}

__device__ void LinearFixBound(float* cord) {
	float temp = *cord;
	if (temp < 0 - FGSHIFT) {
		while (temp < 0 - FGSHIFT) temp += 1;
		*cord = temp;
		return;
	}
	else if (temp > (FGSIZE - 1) - FGSHIFT) {
		while (temp > (FGSIZE - 1) - FGSHIFT) temp -= 1;
		if (temp < 0 - FGSHIFT) {
			temp += 1;
			temp *= -1;
		}
		*cord = temp;
		return;
	}
}

__device__ float LinearInter(struct FuncBase *mas, float x, float y, float z) {
	int i = 0, tempNumNext = -1, tempNumBack = -1;
	float tempX = x, tempY = y, tempZ = z;
	LinearFixBound(&tempX);
	LinearFixBound(&tempY);
	LinearFixBound(&tempZ);
	for (i = 0;i < FGSIZE * FGSIZE * FGSIZE;i++) {
		if (mas[i].x >= tempX && mas[i].y >= tempY && mas[i].z >= tempZ) {
			if (tempNumNext == -1) tempNumNext = i;
			if (mas[i].x <= mas[tempNumNext].x && mas[i].y <= mas[tempNumNext].y && mas[i].z <= mas[tempNumNext].z) tempNumNext = i;
		}
		if (mas[i].x <= tempX && mas[i].y <= tempY && mas[i].z <= tempZ) {
			if (tempNumBack == -1) tempNumBack = i;
			if (mas[i].x >= mas[tempNumBack].x && mas[i].y >= mas[tempNumBack].y && mas[i].z >= mas[tempNumBack].z) tempNumBack = i;
		}
	}

	if (tempNumBack == -1 || tempNumNext == -1) return 0.0;
	float razX = mas[tempNumNext].x - mas[tempNumBack].x;
	float razY = mas[tempNumNext].y - mas[tempNumBack].y;
	float razZ = mas[tempNumNext].z - mas[tempNumBack].z;

	float stepX = tempX - mas[tempNumBack].x;
	float stepY = tempY - mas[tempNumBack].y;
	float stepZ = tempZ - mas[tempNumBack].z;

	float procX = razX == 0 ? 0 : (stepX * 100) / razX;
	float procY = razY == 0 ? 0 : (stepY * 100) / razY;
	float procZ = razZ == 0 ? 0 : (stepZ * 100) / razZ;
	float sumProc = (procX + procY + procZ)/3;

	return mas[tempNumBack].res+((mas[tempNumNext].res - mas[tempNumBack].res) * sumProc) / 100;
}

__global__ void Kernel(float* a, struct Vertex *vertex, struct FuncBase *mas) {
	__shared__ float cache[THREADSPERBLOCK];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid>=VERTCOUNT){
		printf("+");
		cache[threadIdx.x]=0;
		return;
	}
	int cacheIndex = threadIdx.x;
	float x = vertex[tid].x +FGSHIFT + 0.5;
	float y = vertex[tid].y +FGSHIFT + 0.5;
	float z = vertex[tid].z +FGSHIFT + 0.5;
	cache[cacheIndex] = StepInter(mas, x, y, z);
	//cache[cacheIndex] = LinearInter(mas, x, y, z);
	__syncthreads();
	int s = 0;
	for (s = blockDim.x / 2;s > 0;s >>= 1) {
		if (cacheIndex < s) cache[cacheIndex] += cache[cacheIndex + s];
		__syncthreads();
	}
	if (cacheIndex == 0) a[blockIdx.x] = cache[0];
}


int main() {
	struct FuncBase *tempDfH = (struct FuncBase*)malloc(sizeof(struct FuncBase) * FGSIZE * FGSIZE * FGSIZE), *dfH;
	float* mas, *masRes;
	struct Vertex *vertex;

	int i = 0, j = 0, k = 0;
	for (i = 0;i < FGSIZE;i++) {
		for (j = 0;j < FGSIZE;j++) {
			for (k = 0;k < FGSIZE;k++) {
				tempDfH[FGSIZE * (i * FGSIZE + j) + k].res = Function(i - FGSHIFT, j - FGSHIFT, k - FGSHIFT);
				tempDfH[FGSIZE * (i * FGSIZE + j) + k].x = i - FGSHIFT;
				tempDfH[FGSIZE * (i * FGSIZE + j) + k].y = j - FGSHIFT;
				tempDfH[FGSIZE * (i * FGSIZE + j) + k].z = k - FGSHIFT;
			}
		}
	}
	CUDA_CHECK_RETURN(hipMalloc((void**)&vertex, VERTCOUNT * sizeof(struct Vertex)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&mas, FGSIZE * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void**)&dfH, FGSIZE * FGSIZE * FGSIZE * sizeof(struct FuncBase)));
	CUDA_CHECK_RETURN(hipMemcpy(dfH, tempDfH, FGSIZE * FGSIZE * FGSIZE * sizeof(struct FuncBase), hipMemcpyHostToDevice));
	masRes = (float*)malloc(sizeof(float) * BLOCKGRID);

	InitVertices(vertex);
	Kernel <<< dim3(BLOCKGRID), dim3(THREADSPERBLOCK) >>> (mas,vertex,dfH);
	hipDeviceSynchronize();
	hipMemcpy(masRes, mas, BLOCKGRID * sizeof(float), hipMemcpyDeviceToHost);
	float sum = 0;
	for (i = 0;i < BLOCKGRID;i++) sum += masRes[i];
	printf(" Sum = %f\n", sum * M_PI * M_PI / COEF / COEF);
	hipFree(dfH);;
	free(masRes);
	hipFree(vertex);
	free(tempDfH);
	return 0;
}
