#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"

__global__ void InitMas(int *masA){
	masA[threadIdx.x+blockDim.x*blockIdx.x]=threadIdx.x+blockDim.x*blockIdx.x;
}

__global__ void NoShareTran(int *masA, int *masB, int N, int K){
	int curentNum=threadIdx.x+blockDim.x*blockIdx.x;
	int numPos=curentNum/K, numStrok=curentNum%K;
	masB[numPos+numStrok*N]=masA[curentNum];
}

__global__ void ShareError(int *masA, int *masB){
	__shared__ int cacheMas[32][32];
	int k=threadIdx.x+blockIdx.x*blockDim.x;
	int n=threadIdx.y+blockIdx.y*blockDim.y;
	int N=blockDim.x*gridDim.x;
	
	cacheMas[threadIdx.y][threadIdx.x]=masA[k+n*N];
	__syncthreads();
	
	k=threadIdx.x+blockIdx.y*blockDim.x;
	n=threadIdx.y+blockIdx.x*blockDim.y;
	
	masB[k+n*N]=cacheMas[threadIdx.x][threadIdx.y];
}

__global__ void ShareFix(int *masA, int *masB){
	__shared__ int cacheMas[32][33];
	int k=threadIdx.x+blockIdx.x*blockDim.x;
	int n=threadIdx.y+blockIdx.y*blockDim.y;
	int N=blockDim.x*gridDim.x;
	
	cacheMas[threadIdx.y][threadIdx.x]=masA[k+n*N];
	__syncthreads();
	
	k=threadIdx.x+blockIdx.y*blockDim.x;
	n=threadIdx.y+blockIdx.x*blockDim.y;
	
	masB[k+n*N]=cacheMas[threadIdx.x][threadIdx.y];
}

int main(int argc, char **argv){
	if(argc<4){
		fprintf(stdout,"USAGE: ./a <N> <K> <thread_in_block>");
		return 4;
	}
	int nN=atoi(argv[1]), nK=atoi(argv[2]), threadBlock=atoi(argv[3]);
	if((nN*nK)%threadBlock){
		fprintf(stdout,"No del\n");
		return 4;
	}
	
	int *masA, *masB, *masRes, i=0, shag=(nN*nK)/16;
	hipMalloc((void**)&masA,nN*nK*sizeof(int));
	hipMalloc((void**)&masB,nN*nK*sizeof(int));
	masRes=(int*)calloc(nN*nK,sizeof(int));
	if(shag==0) shag=1;
	
	int simpleBlock=(nN*nK)/threadBlock;
	InitMas<<<dim3(simpleBlock),dim3(threadBlock)>>>(masA);
	hipDeviceSynchronize();
	
	NoShareTran<<<dim3(simpleBlock),dim3(threadBlock)>>>(masA,masB,nN,nK);
	hipDeviceSynchronize();
	
	hipMemcpy(masRes,masB,nN*nK*sizeof(int),hipMemcpyDeviceToHost);
	for(i=0;i<nN*nK;i+=shag) fprintf(stdout,"%d ",masRes[i]);
	fprintf(stdout,"\n \n");
	
	int blockNum=sqrt((nN*nK)/(threadBlock*threadBlock));
	if(threadBlock*threadBlock>nN*nK) blockNum=1;
	
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(ShareError),hipFuncCachePreferL1);
	ShareError<<<dim3(blockNum,blockNum),dim3(threadBlock,threadBlock)>>>(masA,masB);
	hipDeviceSynchronize();
	
	hipMemcpy(masRes,masB,nN*nK*sizeof(int),hipMemcpyDeviceToHost);
	for(i=0;i<nN*nK;i+=shag) fprintf(stdout,"%d ",masRes[i]);
	fprintf(stdout,"\n \n");
	
	
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(ShareFix),hipFuncCachePreferL1);
	ShareFix<<<dim3(blockNum,blockNum),dim3(threadBlock,threadBlock)>>>(masA,masB);
	hipDeviceSynchronize();
	
	hipMemcpy(masRes,masB,nN*nK*sizeof(int),hipMemcpyDeviceToHost);
	for(i=0;i<nN*nK;i+=shag) fprintf(stdout,"%d ",masRes[i]);
	fprintf(stdout,"\n \n");
	
	free(masRes);
	hipFree(masA);
	hipFree(masB);
	return 0;
}
