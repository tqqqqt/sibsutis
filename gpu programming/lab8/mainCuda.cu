#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

__global__ void InitMas(float *mas, int masStlb){
	int pos=threadIdx.x+blockIdx.x*blockDim.x;
	int str=threadIdx.y+blockIdx.y*blockDim.y;
	mas[pos+str*masStlb]=(pos+str*masStlb)*0.5;
}

__global__ void MultMas(float *masA, float *masB, float *masRes, int masStr, int masStlb){
	int pos=threadIdx.x+blockIdx.x*blockDim.x;
	int str=threadIdx.y+blockIdx.y*blockDim.y;
	float temp=0;
	int tempPoint=0;
	for(tempPoint=0;tempPoint<masStr;tempPoint+=1) temp+=(float)(masA[tempPoint+str*masStr]*masB[pos+tempPoint*masStlb]);
	masRes[pos+str*masStlb]=temp;
}

__global__ void PrintMas(float *mas, int size, int masStolb){
	int i=0;
	for(i=0;i<size;i+=1){
		printf("%.2f ",mas[i]);
		if((i+1)%masStolb==0) printf("\n");
	}
}

int main(int argc, char **argv){
	if(argc<4){
		fprintf(stdout,"USAGE: ./a <strok> <stolb> <thread_block>\n");
		return 4;
	}
	int strokMas=atoi(argv[1]), stolbMas=atoi(argv[2]), threadBlock=atoi(argv[3]);
	int cudaBlock=sqrt((strokMas*stolbMas)/(threadBlock*threadBlock));
	if((strokMas*stolbMas)<(threadBlock*threadBlock)) cudaBlock=1;
	else if((strokMas*stolbMas)%(threadBlock*threadBlock)!=0){
		fprintf(stdout,"No del\n");
		return 4;
	}
	float *masA, *masB, *masResCuda, *masRes;
	hipMalloc((void**)&masA,strokMas*stolbMas*sizeof(float));
	hipMalloc((void**)&masB,strokMas*stolbMas*sizeof(float));
	hipMalloc((void**)&masResCuda,strokMas*stolbMas*sizeof(float));
	masRes=(float*)malloc(sizeof(float)*strokMas*stolbMas);
	
	
	InitMas<<<dim3(cudaBlock,cudaBlock),dim3(threadBlock,threadBlock)>>>(masA,stolbMas);
	hipDeviceSynchronize();
	PrintMas<<<dim3(1),dim3(1)>>>(masA,(strokMas*stolbMas),stolbMas);
	hipDeviceSynchronize();
	printf("\n \n");
	InitMas<<<dim3(cudaBlock,cudaBlock),dim3(threadBlock,threadBlock)>>>(masB,stolbMas);
	hipDeviceSynchronize();
	PrintMas<<<dim3(1),dim3(1)>>>(masB,(strokMas*stolbMas),stolbMas);
	hipDeviceSynchronize();
	printf("\n \n");
	
	MultMas<<<dim3(cudaBlock,cudaBlock),dim3(threadBlock,threadBlock)>>>(masA,masB,masResCuda,strokMas,stolbMas);
	hipDeviceSynchronize();
	hipMemcpy(masRes,masResCuda,strokMas*stolbMas*sizeof(float),hipMemcpyDeviceToHost);
	int i=0, masAll=strokMas*stolbMas;
	for(i=0;i<masAll;i+=1){
		fprintf(stdout,"%.2f ",masRes[i]);
		if((i+1)%stolbMas==0) fprintf(stdout,"\n");
	}
	
	hipFree(masA);
	hipFree(masB);
	hipFree(masResCuda);
	free(masRes);
	return 0;
}
