#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void InitMas(int *mas){
	mas[threadIdx.x+blockDim.x*blockIdx.x]=threadIdx.x+blockDim.x*blockIdx.x;
}

__global__ void Transparate(int *masA, int *masB, int N, int K){
	int curentNum=threadIdx.x+blockDim.x*blockIdx.x;
	int numPos=curentNum/K, numStrok=curentNum%K;
	masB[numPos+numStrok*N]=masA[curentNum];
}

int main(int argc, char **argv){
	if(argc<5){
		fprintf(stdout,"USAGE: ./a <N> <K> <blocks> <threads>");
		return 4;
	}
	int masN=atoi(argv[1]), masK=atoi(argv[2]), dimBlock=atoi(argv[3]), dimThread=atoi(argv[4]);
	int *masA, *masB, *masRes, i=0;
	hipMalloc((void**)&masA,masN*masK*sizeof(int));
	hipMalloc((void**)&masB,masN*masK*sizeof(int));
	masRes=(int*)malloc(masN*masK*sizeof(int));
	
	InitMas<<<dim3(dimBlock),dim3(dimThread)>>>(masA);
	hipDeviceSynchronize();
	
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	Transparate<<<dim3(dimBlock),dim3(dimThread)>>>(masA,masB,masN,masK);
	hipDeviceSynchronize();
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	
	hipMemcpy(masRes,masB,masN*masK*sizeof(int),hipMemcpyDeviceToHost);
	fprintf(stdout," Time - %g\n",elapsedTime);
	int shag=(masN*masK)/16;
	if(shag<=0) shag=1;
	for(i=0;i<masN*masK;i+=shag) fprintf(stdout,"%d ",masRes[i]);
	
	hipFree(masA);
	hipFree(masB);
	free(masRes);
	return 0;
}
