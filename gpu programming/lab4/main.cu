#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

__global__ void InitMas(int *mas){
	mas[threadIdx.x+blockDim.x*blockIdx.x]=threadIdx.x+blockDim.x*blockIdx.x;
}

__global__ void PreobrazNPars(int *masA, int *masB, int nPars, int nElemPars){
	int curentNum=threadIdx.x+blockDim.x*blockIdx.x;
	int numPos=curentNum/nElemPars, numStrok=curentNum%nElemPars;
	masB[numPos+numStrok*nPars]=masA[curentNum];
}

__global__ void PreobrazBack(int *masA, int *masB, int nPars, int nElemPars){
	int curentNum=threadIdx.x+blockDim.x*blockIdx.x;
	int numPos=curentNum/nElemPars, numStrok=curentNum%nElemPars;
	masB[numStrok+numPos*nElemPars]=masA[curentNum];
}

__global__ void PreobrazXY(int *masA, int *masB, int nPars, int nElemPars){
	int k=threadIdx.x+blockIdx.x*blockDim.x;
	int n=threadIdx.y+blockIdx.y*blockDim.y;
	masB[k+n*nPars]=masA[n+k*nElemPars];
}

__global__ void PreobrazXYBack(int *masA, int *masB, int nPars, int nElemPars){
	int k=threadIdx.x+blockIdx.x*blockDim.x;
	int n=threadIdx.y+blockIdx.y*blockDim.y;
	masB[n+k*nElemPars]=masA[k+n*nPars];
}

int main(int argc, char **argv){
	if(argc<4){
		fprintf(stdout,"Error argums: ./prog <nMas> <nPars> <nElementInPars> \n");
		return 4;
	}
	int nMas=atoi(argv[1]), nPars=atoi(argv[2]), nElemPars=atoi(argv[3]);
	int *masA, *masB, *masRes, i=0;
	masRes=(int*)calloc(nMas,sizeof(int));
	hipMalloc((void**)&masA,nMas*sizeof(int));
	hipMalloc((void**)&masB,nMas*sizeof(int));
	
	fprintf(stdout,"Mas A:\n");
	for(i=0;i<nMas;i++){
		masRes[i]=i;
		fprintf(stdout,"%d ",masRes[i]);
		masRes[i]=0;
	}
	fprintf(stdout,"\n \n");
	
	InitMas<<<dim3(nMas),dim3(1)>>>(masA);
	hipDeviceSynchronize();
	PreobrazNPars<<<dim3(nPars),dim3(nElemPars)>>>(masA,masB,nPars,nElemPars);
	hipDeviceSynchronize();
		
	hipMemcpy(masRes,masB,nMas*sizeof(int),hipMemcpyDeviceToHost);
	fprintf(stdout,"Preobraz 1:\n");
	for(i=0;i<nMas;i++) fprintf(stdout,"%d ",masRes[i]);
	fprintf(stdout,"\n \n");
	
	for(i=0;i<nMas;i++) masRes[i]=0;
	PreobrazBack<<<dim3(nPars),dim3(nElemPars)>>>(masA,masB,nPars,nElemPars);
	hipDeviceSynchronize();
	
	hipMemcpy(masRes,masB,nMas*sizeof(int),hipMemcpyDeviceToHost);
	fprintf(stdout,"Preobraz back:\n");
	for(i=0;i<nMas;i++) fprintf(stdout,"%d ",masRes[i]);
	fprintf(stdout,"\n \n");
	
	for(i=0;i<nMas;i++) masRes[i]=0;
	PreobrazXY<<<dim3(nPars,nElemPars),dim3(1,1)>>>(masA,masB,nPars,nElemPars);
	hipDeviceSynchronize();
	
	hipMemcpy(masRes,masB,nMas*sizeof(int),hipMemcpyDeviceToHost);
	fprintf(stdout,"Preobraz XY 1:\n");
	for(i=0;i<nMas;i++){
		fprintf(stdout,"%d ",masRes[i]);
		if((i+1)%nPars==0) fprintf(stdout,"\n");
	}
	fprintf(stdout,"\n \n");
	
	for(i=0;i<nMas;i++) masRes[i]=0;
	PreobrazXYBack<<<dim3(nPars,nElemPars),dim3(1,1)>>>(masA,masB,nPars,nElemPars);
	hipDeviceSynchronize();
	
	hipMemcpy(masRes,masB,nMas*sizeof(int),hipMemcpyDeviceToHost);
	fprintf(stdout,"Preobraz XY 2:\n");
	for(i=0;i<nMas;i++){
		fprintf(stdout,"%d ",masRes[i]);
		if((i+1)%nElemPars==0) fprintf(stdout,"\n");
	}
	fprintf(stdout,"\n \n");
	
	free(masRes);
	hipFree(masA);
	hipFree(masB);
	return 0;
}
