#include <stdio.h>
#include <hip/hip_runtime.h>

int main(){
	hipDeviceProp_t prop;
	int count;
	hipGetErrorString(hipGetDeviceCount(&count));
	for(int i=0;i<count;i+=1){
		hipGetErrorString(hipGetDeviceProperties(&prop,i));
		printf("----info %d device----\n",i);
		printf("Name: %s\n",prop.name);
		printf("Vichslit vozmoz: %d.%d\n",prop.major,prop.minor);
		printf("Chastota: %d\n",prop.clockRate);
		printf("Perecritie: ");
		if(prop.deviceOverlap) printf("Yes\n");
		else printf("No\n");
		printf("Time-out thread: ");
		if(prop.kernelExecTimeoutEnabled) printf("Yes\n");
		else printf("No\n");
		
		printf("\n----info memory device %d----\n",i);
		printf("All global mem: %zd\n",prop.totalGlobalMem);
		printf("All const mem: %zd\n",prop.totalConstMem);
		printf("Max step: %zd\n",prop.memPitch);
		printf("Fix texture: %zd\n",prop.textureAlignment);
		
		printf("\n----info multi-proc device %d----\n",i);
		printf("Count multi-proc: %d\n",prop.multiProcessorCount);
		printf("Shared mem on 1 MP: %zd\n",prop.sharedMemPerBlock);
		printf("Registr on 1 MP: %d\n",prop.regsPerBlock);
		printf("Thread in warp: %d\n",prop.warpSize);
		printf("Max thread in block: %d\n",prop.maxThreadsPerBlock);
		printf("Thread in dim: (%d, %d, %d)\n",prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2]);
		printf("Max size grid: (%d, %d, %d)\n",prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
		printf("\n");
	}
}
