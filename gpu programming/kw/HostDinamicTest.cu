#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N_SIZE 1024
#define OFSET 1024
#define FULL_DATA ((N_SIZE*N_SIZE)*20)

__global__ void Kernel(double *masA, double *masB, int endPoint, int curentPoint){
	int curentIndex=threadIdx.x+blockDim.x*blockIdx.x;
	if(curentIndex<endPoint){
		double temp=0;
		for(int j=0.0;j<endPoint;j+=1) temp+=((sin(3.141592)*j*masA[j]*curentPoint)*0.01)*0.01;
		masB[curentIndex]=temp;
	}
}

int main(int argc, char** argv){
	if(argc!=2){
		printf("USAGE: ./a <count_streams>\n");
		return -4;
	}
	int countDevice=0;
	hipGetDeviceCount(&countDevice);
	if(countDevice<1){
		printf("No GPU device.\n");
		return -4;
	}
	int inputStream=atoi(argv[1]);
	if(inputStream<=0){
		printf("Need streams >= 1\n");
		return -4;
	}
	printf("System have %d devices.\n",countDevice);
	double **deviceA, **deviceB;
	double *hostA, *hostC;
	deviceA=(double**)calloc((FULL_DATA/OFSET)+1,sizeof(double*));
	deviceB=(double**)calloc((FULL_DATA/OFSET)+1,sizeof(double*));
	hipHostMalloc((void**)&hostA,FULL_DATA*sizeof(double), hipHostMallocDefault);
	hipHostMalloc((void**)&hostC,FULL_DATA*sizeof(double), hipHostMallocDefault);
	hipStream_t *stream;
	stream=(hipStream_t*)malloc(inputStream*sizeof(hipStream_t));
	for(int i=0;i<inputStream;i+=1) hipStreamCreateWithFlags(&stream[i],hipStreamNonBlocking);
	
	for(int i=0;i<FULL_DATA;i+=1){
		hostC[i]=0;
		hostA[i]=i;
	}
	
	
	int block=(OFSET/256), thread=256, curentMas=0;
	if(block<=0) block=1;
	if(OFSET<256) thread=OFSET;
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	for(int i=0;i<FULL_DATA;i+=0){
		for(int j=0;j<inputStream && i<FULL_DATA;j+=1, i+=OFSET){
			hipMalloc((void**)&deviceA[curentMas],OFSET*sizeof(double));
			hipMalloc((void**)&deviceB[curentMas],OFSET*sizeof(double));
			hipMemcpyAsync(deviceA[curentMas],hostA+i,OFSET*sizeof(double),hipMemcpyHostToDevice,stream[j]);
			Kernel<<<dim3(block),dim3(thread),0,stream[j]>>>(deviceA[curentMas],deviceB[curentMas],OFSET,i);
			hipMemcpyAsync(hostC+i,deviceB[curentMas],OFSET*sizeof(double),hipMemcpyDeviceToHost,stream[j]);
		}
	}
	for(int i=0;i<inputStream;i+=1) hipStreamSynchronize(stream[i]);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf(" Time - %g\n",elapsedTime);
	
	int shag=FULL_DATA/16;
	if(shag<=0) shag=1;
	for(int i=0;i<FULL_DATA;i+=shag) printf("%d) %f\n",i,hostC[i]);
	
	for(int i=0;i<inputStream;i+=1) hipStreamDestroy(stream[i]);
	hipFree(deviceA);
	hipFree(deviceB);
	hipHostFree(hostA);
	hipHostFree(hostC);
	return 0;
}
