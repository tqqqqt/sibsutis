#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void InitMas(float *masA, float *masB, int &N){
	int curentIndex=threadIdx.x+blockDim.x*blockIdx.x;
	masA[curentIndex]=curentIndex;
	masB[curentIndex]=0.5;
}

__global__ void MultiMas(float *masA, float *masB, float *masRes){
	int curentIndex=threadIdx.x+blockDim.x*blockIdx.x;
	masRes[curentIndex]=masA[curentIndex]*masB[curentIndex];
}

int main(int argc, char **argv){
	if(argc<4){
		fprintf(stdout,"USAGE: ./a <N> <dimBlock> <dimThread>");
		return 4;
	}
	int masN=atoi(argv[1]), dimThread=atoi(argv[2]), dimBlock=atoi(argv[3]);
	float *masA, *masB, *masCudaRes, *masRes, sum=0;
	int i=0;
	hipMalloc((void**)&masA,masN*sizeof(float));
	hipMalloc((void**)&masB,masN*sizeof(float));
	hipMalloc((void**)&masCudaRes,masN*sizeof(float));
	masRes=(float*)malloc(masN*sizeof(float));
	InitMas<<<dim3(dimBlock),dim3(dimThread)>>>(masA,masB,masN);
	hipDeviceSynchronize();
	
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	MultiMas<<<dim3(dimBlock),dim3(dimThread)>>>(masA,masB,masCudaRes);
	hipDeviceSynchronize();
	hipMemcpy(masRes,masCudaRes,masN*sizeof(float),hipMemcpyDeviceToHost);
	for(i=0;i<masN;i++) sum+=masRes[i];
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	
	fprintf(stdout," Time - %g\n Sum - %g\n",elapsedTime,sum);
	hipFree(masA);
	hipFree(masB);
	hipFree(masCudaRes);
	free(masRes);
	return 0;
}
