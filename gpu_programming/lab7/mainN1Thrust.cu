#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/inner_product.h>
#include <cstdio>

int main(int argc, char **argv){
	if(argc<2){
		fprintf(stdout,"USAGE: ./a <N>");
		return 4;
	}
	int N=atoi(argv[1]);
	float sum=0;
	
	thrust::host_vector<float> h_V(N);
	thrust::device_vector<float> d_V1(N);
	thrust::device_vector<float> d_V2(N);
	thrust::device_vector<float> d_V3(N);
	
	thrust::sequence(d_V1.begin(),d_V1.end());
	thrust::fill(d_V2.begin(),d_V2.end(),0.5);
	
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	thrust::transform(d_V1.begin(),d_V1.end(),d_V2.begin(),d_V3.begin(),thrust::multiplies<float>());
	h_V=d_V3;
	sum=thrust::reduce(h_V.begin(),h_V.end(),0.0,thrust::plus<float>());
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	
	fprintf(stdout," Time - %g\n Sum - %g\n",elapsedTime,sum);
	return 0;
}
