#include <stdio.h>
#include <thrust/generate.h>
#include <thrust/gather.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

int main(int argc, char **argv){
	if(argc<3){
		fprintf(stdout,"USAGE: ./a <N> <K>\n");
		return 4;
	}
	int masN=atoi(argv[1]), masK=atoi(argv[2]), i=0;
	//thrust::host_vector<int> h_A(masN*masK);
	thrust::host_vector<int> h_AT(masN*masK);
	thrust::device_vector<int> d_A(masN*masK);
	thrust::device_vector<int> d_AT(masN*masK);
	thrust::sequence(d_A.begin(),d_A.end());
	
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	int *map=(int*)malloc(masN*masK*sizeof(int));
	for(i=0;i<masN*masK;i++) map[i]=(i%masN)*masK+(i/masN);
	thrust::device_vector<int> d_map(map,map+masK*masN);
	thrust::gather(d_map.begin(),d_map.end(),d_A.begin(),d_AT.begin());
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	
	h_AT=d_AT;
	fprintf(stdout," Time - %g\n",elapsedTime);
	int shag=(masN*masK)/16;
	if(shag<=0) shag=1;
	for(i=0;i<masN*masK;i+=shag) fprintf(stdout,"%d ",h_AT[i]);
	free(map);
	return 0;
}
