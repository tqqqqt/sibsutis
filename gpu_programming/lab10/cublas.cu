#pragma comment (lib, "cublas.lib")


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "hipblas.h"

__global__ void PrintMas(float *mas, int size, int masStolb){
	int i=0;
	for(i=0;i<size;i+=1){
		printf("%.2f ",mas[i]);
		if((i+1)%masStolb==0) printf("\n");
	}
}

int main(int argc, char **argv){
	if(argc!=3){
		fprintf(stdout,"USAGE: ./a <strok> <stolb>\n");
		return 4;
	}
	int strokMas=atoi(argv[1]), stolbMas=atoi(argv[2]);
	int elements=strokMas*stolbMas;
	const size_t sizeInBytes=(elements*sizeof(float));
	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);
	
	float *masADev, *masAHost, *masBDev, *masBHost, *masCDev;
	hipMalloc((void**)&masADev,sizeInBytes);
	hipMalloc((void**)&masBDev,sizeInBytes);
	hipMalloc((void**)&masCDev,sizeInBytes);
	hipHostMalloc((void**)&masAHost,sizeInBytes, hipHostMallocDefault);
	hipHostMalloc((void**)&masBHost,sizeInBytes, hipHostMallocDefault);
	
	memset(masAHost,0,sizeInBytes);
	memset(masBHost,0,sizeInBytes);

	
	hipblasSetMatrix(strokMas,stolbMas,sizeof(*masAHost),masAHost,strokMas,masADev,strokMas);
	hipblasSetMatrix(strokMas,stolbMas,sizeof(*masAHost),masBHost,strokMas,masBDev,strokMas);
	const float alpha=1.0;
	const float beta=0.0;
	hipblasSgemm(cublas_handle,HIPBLAS_OP_N,HIPBLAS_OP_N,strokMas,stolbMas,strokMas,&alpha,masADev,strokMas,masBDev,stolbMas,&beta,masCDev,strokMas);
	hipblasGetMatrix(strokMas,stolbMas,sizeof(float),masCDev,strokMas,masAHost,strokMas);
	
	hipblasDestroy(cublas_handle);
	hipFree(masADev);
	hipFree(masBDev);
	hipFree(masCDev);
	hipHostFree(masAHost);
	hipHostFree(masBHost);
	return 0;
}
