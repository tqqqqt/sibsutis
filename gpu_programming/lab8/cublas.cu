#pragma comment (lib, "cublas.lib")

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

__global__ void PrintMas(float *mas, int size, int masStolb){
	int i=0;
	for(i=0;i<size;i+=1){
		printf("%.2f ",mas[i]);
		if((i+1)%masStolb==0) printf("\n");
	}
}

int main(int argc, char **argv){
	if(argc!=3){
		fprintf(stdout,"USAGE: ./a <strok> <stolb>\n");
		return 4;
	}
	int strokMas=atoi(argv[1]), stolbMas=atoi(argv[2]);
	int elements=strokMas*stolbMas;
	const size_t sizeInBytes=(elements*sizeof(float));
	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);
	
	float *masADev, *masAHost, *masBDev, *masBHost, *masCDev;
	hipMalloc((void**)&masADev,sizeInBytes);
	hipMalloc((void**)&masBDev,sizeInBytes);
	hipMalloc((void**)&masCDev,sizeInBytes);
	hipHostMalloc((void**)&masAHost,sizeInBytes, hipHostMallocDefault);
	hipHostMalloc((void**)&masBHost,sizeInBytes, hipHostMallocDefault);
	
	memset(masAHost,0,sizeInBytes);
	memset(masBHost,0,sizeInBytes);
	
	int i=0, j=0;
	for(i=0;i<strokMas;i+=1){
		for(j=0;j<stolbMas;j+=1){
			masAHost[j+i*stolbMas]=(j+i*stolbMas)*0.5;
			masBHost[j+i*stolbMas]=(j+i*stolbMas)*0.5;
		}
	}
	
	
	hipblasSetMatrix(strokMas,stolbMas,sizeof(*masAHost),masAHost,strokMas,masADev,strokMas);
	hipblasSetMatrix(strokMas,stolbMas,sizeof(*masAHost),masBHost,strokMas,masBDev,strokMas);
	PrintMas<<<dim3(1),dim3(1)>>>(masADev,elements,stolbMas);
	hipDeviceSynchronize();
	fprintf(stdout,"\n \n");
	PrintMas<<<dim3(1),dim3(1)>>>(masBDev,elements,stolbMas);
	hipDeviceSynchronize();
	fprintf(stdout,"\n \n");
	const float alpha=1.0;
	const float beta=0.0;
	hipblasSgemm(cublas_handle,HIPBLAS_OP_N,HIPBLAS_OP_N,strokMas,stolbMas,strokMas,&alpha,masADev,strokMas,masBDev,stolbMas,&beta,masCDev,strokMas);
	hipblasGetMatrix(strokMas,stolbMas,sizeof(float),masCDev,strokMas,masAHost,strokMas);
	
	for(i=0;i<elements;i+=1){
		fprintf(stdout,"%.2f ",masAHost[i]);
		if((i+1)%stolbMas==0) fprintf(stdout,"\n");
	}
	hipblasDestroy(cublas_handle);
	hipFree(masADev);
	hipFree(masBDev);
	hipFree(masCDev);
	hipHostFree(masAHost);
	hipHostFree(masBHost);
	return 0;
}
