#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N_SIZE 1024
#define OFSET 1024
#define FULL_DATA ((N_SIZE*N_SIZE)*20)

__global__ void Kernel(double *masA, double *masB, int endPoint, int curentPoint){
	int curentIndex=threadIdx.x+blockDim.x*blockIdx.x;
	if(curentIndex<endPoint){
		double temp=0;
		for(int j=0.0;j<endPoint;j+=1) temp+=((sin(3.141592)*j*masA[j]*curentPoint)*0.01)*0.01;
		masB[curentIndex]=temp;
	}
}

int main(int argc, char** argv){
	if(argc!=2){
		printf("USAGE: ./a <num_streams>\n");
		return -4;
	}
	int countDevice=0;
	hipGetDeviceCount(&countDevice);
	if(countDevice<1){
		printf("No GPU device.\n");
		return -4;
	}
	int inputCountStream=atoi(argv[1]);
	if(inputCountStream<=0){
		printf("Streams need > 0\n");
		return -4;
	}
	printf("System have %d devices.\n",countDevice);
	double **deviceA, **deviceB, **deviceHostA, **deviceHostC;
	double *hostC;
	int trigerPerDevice=(FULL_DATA/OFSET)+1;
	deviceA=(double**)calloc(trigerPerDevice,sizeof(double*));
	deviceB=(double**)calloc(trigerPerDevice,sizeof(double*));
	deviceHostA=(double**)calloc(trigerPerDevice,sizeof(double*));
	deviceHostC=(double**)calloc(trigerPerDevice,sizeof(double*));
	hostC=(double*)malloc(FULL_DATA*sizeof(double));
	hipStream_t *stream;
	stream=(hipStream_t*)calloc(countDevice*inputCountStream,sizeof(hipStream_t));
	for(int i=0;i<countDevice;i+=1){
		hipSetDevice(i);
		for(int j=0;j<inputCountStream;j+=1) hipStreamCreateWithFlags(&stream[i*inputCountStream+j],hipStreamNonBlocking);
	}	
	for(int i=0;i<FULL_DATA;i+=1) hostC[i]=0;
	int curentDevice=0, curentMas=0;
	for(int i=0;i<FULL_DATA;i+=0){
		hipSetDevice(curentDevice);
		for(int j=0;j<inputCountStream && i<FULL_DATA;j+=1){
			hipHostMalloc((void**)&deviceHostA[curentMas],OFSET*sizeof(double), hipHostMallocDefault);
			hipHostMalloc((void**)&deviceHostC[curentMas],OFSET*sizeof(double), hipHostMallocDefault);
			for(int k=0;k<OFSET && i<FULL_DATA;k+=1, i+=1) deviceHostA[curentMas][k]=i;
			curentMas+=1;
		}
		curentDevice+=1;
		if(curentDevice>=countDevice) curentDevice=0;
	}
	
	
	curentDevice=0;
	curentMas=0;
	int block=(OFSET/256), thread=256;
	if(block<=0) block=1;
	if(OFSET<256) thread=OFSET;
	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	for(int i=0;i<FULL_DATA;i+=0){
		hipSetDevice(curentDevice);
		for(int j=0;j<inputCountStream && i<FULL_DATA;j+=1, i+=OFSET){
			hipMalloc((void**)&deviceA[curentMas],OFSET*sizeof(double));
			hipMalloc((void**)&deviceB[curentMas],OFSET*sizeof(double));
			hipMemcpyAsync(deviceA[curentMas],deviceHostA[curentMas],OFSET*sizeof(double),hipMemcpyHostToDevice,stream[curentDevice*inputCountStream+j]);
			Kernel<<<dim3(block),dim3(thread),0,stream[curentDevice*inputCountStream+j]>>>(deviceA[curentMas],deviceB[curentMas],OFSET,i);
			hipMemcpyAsync(deviceHostC[curentMas],deviceB[curentMas],OFSET*sizeof(double),hipMemcpyDeviceToHost,stream[curentDevice*inputCountStream+j]);
			curentMas+=1;
		}
		curentDevice+=1;
		if(curentDevice>=countDevice) curentDevice=0;
	}
	for(int i=0;i<countDevice;i+=1){
		hipSetDevice(i);
		for(int j=0;j<inputCountStream;j+=1) hipStreamSynchronize(stream[i*inputCountStream+j]);
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf(" Time - %g\n",elapsedTime);
	
	curentDevice=0;
	curentMas=0;
	for(int i=0;i<FULL_DATA;i+=0){
		hipSetDevice(curentDevice);
		for(int j=0;j<inputCountStream && i<FULL_DATA;j+=1, i+=OFSET){
			hipMemcpy(hostC+i,deviceHostC[curentMas],OFSET*sizeof(double),hipMemcpyDeviceToHost);
			curentMas+=1;
		}
		curentDevice+=1;
		if(curentDevice>=countDevice) curentDevice=0;
	}

	int shag=FULL_DATA/16;
	if(shag<=0) shag=1;
	for(int i=0;i<FULL_DATA;i+=shag) printf("%d) %f\n",i,hostC[i]);
	
	for(int i=0;i<countDevice;i+=1){
		hipSetDevice(i);
		for(int j=0;j<inputCountStream;j+=1) hipStreamDestroy(stream[i*inputCountStream+j]);
	}
	hipFree(deviceA);
	hipFree(deviceB);
	hipHostFree(deviceHostA);
	hipHostFree(deviceHostC);
	free(hostC);
	return 0;
}
