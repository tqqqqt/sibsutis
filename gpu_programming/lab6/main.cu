#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define COEF 48
#define VERTCOUNT COEF*COEF*2
#define M_PI 3.141592
#define RADIUS 10.0f
#define FGSIZE 20
#define FGSHIFT FGSIZE/2
#define THREADSPERBLOCK 256
#define IMIN(A,B) (A<B?A:B)
#define BLOCKGRID IMIN(32,(VERTCOUNT+THREADSPERBLOCK-1)/THREADSPERBLOCK)

struct Vertex {
	float x, y, z;
};

__constant__ Vertex vert[VERTCOUNT];
hipArray* dfArray = 0;
texture<float, 3, hipReadModeElementType> dfTex;

float Function(float vx, float vy, float vz) {
	return (0.5 * sqrtf(15.0 / M_PI)) * (0.5 * sqrtf(15.0 / M_PI)) * vz * vz * vy * vy * sqrtf(1.0 - vz * vz / RADIUS / RADIUS) / RADIUS / RADIUS / RADIUS / RADIUS;
}

float check(Vertex* v) {
	float sum = 0;
	int i = 0;
	for (i = 0;i < VERTCOUNT;i++) {
		sum += Function(v[i].x,v[i].y,v[i].z);
	}
	return sum;
}

void InitVertices() {
	struct Vertex *tempVert = (struct Vertex*)malloc(VERTCOUNT * sizeof(struct Vertex));
	int i = 0, iphi = 0, ipsi = 0;
	for (iphi = 0;iphi < 2 * COEF;++iphi) {
		for (ipsi = 0;ipsi < COEF;++ipsi, ++i) {
			float phi = iphi * M_PI / COEF;
			float psi = ipsi * M_PI / COEF;
			tempVert[i].x = RADIUS * sinf(psi) * cosf(phi);
			tempVert[i].y = RADIUS * sinf(psi) * sinf(phi);
			tempVert[i].z = RADIUS * cosf(psi);
		}
	}
	printf("\n Sumcheck = %f\n", check(tempVert) * M_PI * M_PI / COEF / COEF);
	hipMemcpyToSymbol(HIP_SYMBOL(vert), tempVert, sizeof(Vertex) * VERTCOUNT, 0, hipMemcpyHostToDevice);
	free(tempVert);
}

void LoadTexture(float* dfH) {
	const hipExtent volumeSize = make_hipExtent(FGSHIFT,FGSHIFT,FGSHIFT);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMalloc3DArray(&dfArray, &channelDesc, volumeSize);

	hipMemcpy3DParms cpyParams = { 0 };
	cpyParams.srcPtr = make_hipPitchedPtr((void*)dfH, volumeSize.width * sizeof(float), volumeSize.width, volumeSize.height);
	cpyParams.dstArray = dfArray;
	cpyParams.extent = volumeSize;
	cpyParams.kind = hipMemcpyHostToDevice;

	hipMemcpy3D(&cpyParams);
}

void TuneTexture(hipChannelFormatDesc channelDesc) {
	dfTex.normalized = false;
	dfTex.filterMode = hipFilterModeLinear;
	dfTex.addressMode[0] = hipAddressModeClamp;
	dfTex.addressMode[1] = hipAddressModeClamp;
	dfTex.addressMode[2] = hipAddressModeClamp;

	hipBindTextureToArray(dfTex, dfArray, channelDesc);
}

__global__ void Kernel(float* a) {
	__shared__ float cache[THREADSPERBLOCK];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;
	float x = vert[tid].x +FGSHIFT + 0.5;
	float y = vert[tid].y +FGSHIFT + 0.5;
	float z = vert[tid].z +FGSHIFT + 0.5;
	cache[cacheIndex] = tex3D(dfTex, z, y, x);
	__syncthreads();
	int s = 0;
	for (s = blockDim.x / 2;s > 0;s >>= 1) {
		if (cacheIndex < s) cache[cacheIndex] += cache[cacheIndex + s];
		__syncthreads();
	}
	if (cacheIndex == 0) a[blockIdx.x] = cache[0];
}


void ReleaseTexture() {
	hipUnbindTexture(dfTex);
	hipFreeArray(dfArray);
}

int main() {
	float* dfH = (float*)malloc(sizeof(float) * FGSIZE * FGSIZE * FGSIZE);
	float* mas, *masRes;
	int i = 0, j = 0, k = 0;
	for (i = 0;i < FGSIZE;i++) {
		for (j = 0;j < FGSIZE;j++) {
			for (k = 0;k < FGSIZE;k++) {
				dfH[FGSIZE * (i * FGSIZE + j) + k] = Function(i - FGSHIFT, j - FGSHIFT, k - FGSHIFT);
			}
		}
	}
	hipMalloc((void**)&mas, FGSIZE * sizeof(float));
	masRes = (float*)malloc(sizeof(float) * BLOCKGRID);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	InitVertices();
	LoadTexture(dfH);
	TuneTexture(channelDesc);
	Kernel <<< dim3(BLOCKGRID), dim3(THREADSPERBLOCK) >>> (mas);
	hipDeviceSynchronize();
	hipMemcpy(masRes, mas, BLOCKGRID * sizeof(float), hipMemcpyDeviceToHost);
	float sum = 0;
	for (i = 0;i < BLOCKGRID;i++) sum += masRes[i];
	printf(" Sum = %f\n", sum * M_PI * M_PI / COEF / COEF);
	ReleaseTexture();
	free(dfH);
	hipFree(mas);
	free(masRes);
	return 0;
}
