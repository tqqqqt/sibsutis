#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK_RETURN(value) {\
	hipError_t _m_cudaStat=value;\
	if(_m_cudaStat!=hipSuccess){\
		fprintf(stderr,"Error %s at line %d in file %s\n",hipGetErrorString(_m_cudaStat),__LINE__,__FILE__);\
		exit(1);\
	} }

__global__ void InitDeviceMas(float *mas, int N){
	int temp=threadIdx.x+blockIdx.x*blockDim.x;
	if(temp<N) mas[temp]=temp*temp;
}

__global__ void ClearDeviceMas(float *mas, int N){
	int temp=threadIdx.x+blockIdx.x*blockDim.x;
	if(temp<N) mas[temp]=0;
}

__global__ void PrintDeviceMas(float *mas, int N, int shag){
	printf("\n");
	for(int i=0;i<N;i+=shag) printf("%.2f ",mas[i]);
	printf("\n");
}

void InitHostMas(float *mas, int N){
	for(int i=0;i<N;i+=1) mas[i]=i*i;
}

void ClearHostMas(float *mas, int N){
	for(int i=0;i<N;i+=1) mas[i]=0;
}

void PrintHostMas(float *mas, int N, int shag){
	printf("\n");
	for(int i=0;i<N;i+=shag) printf("%.2f ",mas[i]);
	printf("\n");
}

int main(int argc, char **argv){
	if(argc!=2){
		fprintf(stderr,"USAGE: ./a <N>\n");
		return -1;
	}
	int N=atoi(argv[1]), shag=N/16, blockDim=N/1024;
	if(shag==0) shag=1;
	if(blockDim==0) blockDim=1;
	float *masHost1, *masHost2, *masDevice;
	float elapsedTime;
	hipEvent_t start, stop;
	
	masHost1=(float*)malloc(sizeof(float)*N);
	CUDA_CHECK_RETURN(hipHostMalloc((void**)&masHost2,sizeof(float)*N, hipHostMallocDefault));
	CUDA_CHECK_RETURN(hipMalloc((void**)&masDevice,sizeof(float)*N));
	
	InitHostMas(masHost1,N);
	InitHostMas(masHost2,N);
	ClearDeviceMas<<<dim3(blockDim),dim3(1024)>>>(masDevice,N);
	hipDeviceSynchronize();
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	hipMemcpy(masDevice,masHost1,N*sizeof(float),hipMemcpyHostToDevice);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("malloc HtoD: %g\n",elapsedTime);
	PrintDeviceMas<<<dim3(1),dim3(1)>>>(masDevice,N,shag);
	hipDeviceSynchronize();
	
	ClearDeviceMas<<<dim3(blockDim),dim3(1024)>>>(masDevice,N);
	hipDeviceSynchronize();
	printf("\n------------------\n");
	hipEventRecord(start,0);
	hipMemcpy(masDevice,masHost2,N*sizeof(float),hipMemcpyHostToDevice);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("hipHostMalloc HtoD: %g\n",elapsedTime);
	PrintDeviceMas<<<dim3(1),dim3(1)>>>(masDevice,N,shag);
	hipDeviceSynchronize();
	
	ClearDeviceMas<<<dim3(blockDim),dim3(1024)>>>(masDevice,N);
	hipDeviceSynchronize();
	printf("\n \n------------------\n");
	hipEventRecord(start,0);
	hipMemcpy(masHost1,masDevice,N*sizeof(float),hipMemcpyDeviceToHost);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("malloc DtoH: %g\n",elapsedTime);
	PrintHostMas(masHost1,N,shag);	
	
	printf("\n------------------\n");
	hipEventRecord(start,0);
	hipMemcpy(masHost2,masDevice,N*sizeof(float),hipMemcpyDeviceToHost);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	printf("hipHostMalloc DtoH: %g\n",elapsedTime);
	PrintHostMas(masHost2,N,shag);
	
	
	hipHostFree(masHost2);
	hipFree(masDevice);
	free(masHost1);
	return 0;
}
