#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define ST 1024
#define N (ST*ST)
#define FULL_DATA (N*20)

#define CUDA_CHECK_RETURN(value) {\
	hipError_t _m_cudaStat=value;\
	if(_m_cudaStat!=hipSuccess){\
		fprintf(stderr,"Error %s at line %d in file %s\n",hipGetErrorString(_m_cudaStat),__LINE__,__FILE__);\
		exit(1);\
	} }

__global__ void MultiMas(float *masA, float *masB, float *masRes){
	int temp=threadIdx.x+blockIdx.x*blockDim.x;
	if(temp<(N*2)) masRes[temp]=masA[temp]*masB[temp];
}

int main(){
	hipDeviceProp_t prop;
	int whichDevice;
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop,whichDevice);
	if(!prop.deviceOverlap){
		printf("Device does not support overlapping\n");
		return -1;
	}
	float *hostA, *hostB, *hostC;
	float *deviceA, *deviceB, *deviceC;
	hipMalloc((void**)&deviceA,(N*2)*sizeof(float));
	hipMalloc((void**)&deviceB,(N*2)*sizeof(float));
	hipMalloc((void**)&deviceC,(N*2)*sizeof(float));
	hipHostMalloc((void**)&hostA,FULL_DATA*sizeof(float));
	hipHostMalloc((void**)&hostB,FULL_DATA*sizeof(float));
	hipHostMalloc((void**)&hostC,FULL_DATA*sizeof(float));
	for(int i=0;i<FULL_DATA;i+=1){
		hostA[i]=i;
		hostB[i]=0.5;
	}
	hipStream_t stream[2];
	for(int i=0;i<2;i+=1) hipStreamCreate(&stream[i]);
	
	int block=((N*2)/256);
	if(block==0) block=1;
	size_t curStream=1;
	for(int i=0;i<FULL_DATA;i+=(N*2)){
		curStream=1-curStream;
		hipMemcpyAsync(deviceA,hostA+i,(N*2)*sizeof(float),hipMemcpyHostToDevice,stream[curStream]);
		hipMemcpyAsync(deviceB,hostB+i,(N*2)*sizeof(float),hipMemcpyHostToDevice,stream[curStream]);
		MultiMas<<<dim3(block),dim3(256),0,stream[curStream]>>>(deviceA,deviceB,deviceC);
		hipMemcpyAsync(hostC+i,deviceC,(N*2)*sizeof(float),hipMemcpyDeviceToHost,stream[curStream]);
	}
	for(int i=0;i<2;i+=1) hipStreamSynchronize(stream[i]);
	
	
	float res=0;
	for(int i=0;i<FULL_DATA;i+=1) res+=hostC[i];
	printf("Sum = %.2f\n",res);

	for(int i=0;i<2;i+=1) hipStreamDestroy(stream[i]);
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	hipHostFree(hostA);
	hipHostFree(hostB);
	hipHostFree(hostC);
	return 0;
}
